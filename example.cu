#include "tensor-enum.cuh"


int main(){
    Tensor<float> a = {NULL, {2, 3, 4}};
    Tensor<float> b = {NULL, {2, 3, 4}}; 

    hipMalloc(&a.data, get_size(a.shape) * sizeof(float));
    hipMalloc(&b.data, get_size(b.shape) * sizeof(float));



    hipFree(a.data);
    hipFree(b.data);
    return 0;
}